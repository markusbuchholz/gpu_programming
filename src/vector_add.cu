#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>   
#include <ctime>     

// CUDA kernel for vector addition
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main() {
    srand(static_cast<unsigned>(time(0)));

    int N = 1 << 20; // 1000K elements
    size_t size = N * sizeof(float);

    // allocate host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // allocate device memory
    float *d_A, *d_B, *d_C;
    hipError_t err;

    err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device memory for A (error code " 
                  << hipGetErrorString(err) << ")!" << std::endl;
        return EXIT_FAILURE;
    }

    err = hipMalloc((void**)&d_B, size);
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device memory for B (error code " 
                  << hipGetErrorString(err) << ")!" << std::endl;
        hipFree(d_A);
        return EXIT_FAILURE;
    }

    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device memory for C (error code " 
                  << hipGetErrorString(err) << ")!" << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        return EXIT_FAILURE;
    }

    // copy data from host to device
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Failed to copy A from host to device (error code " 
                  << hipGetErrorString(err) << ")!" << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return EXIT_FAILURE;
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Failed to copy B from host to device (error code " 
                  << hipGetErrorString(err) << ")!" << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return EXIT_FAILURE;
    }

    // launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // check for any errors launching the kernel
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Failed to launch vectorAdd kernel (error code " 
                  << hipGetErrorString(err) << ")!" << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return EXIT_FAILURE;
    }

    // copy result back to host
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Failed to copy C from device to host (error code " 
                  << hipGetErrorString(err) << ")!" << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return EXIT_FAILURE;
    }

    // Checks
    bool success = true;
    for (int i = 0; i < N; ++i) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            std::cerr << "Mismatch at index " << i << ": " << h_C[i]
                      << " != " << h_A[i] + h_B[i] << std::endl;
            success = false;
            break;
        }
    }

    if (success) {
        std::cout << "Vector addition successful!" << std::endl;
    } else {
        std::cout << "Vector addition failed!" << std::endl;
    }

    std::cout << "\nFirst 10 elements of each vector:" << std::endl;
    std::cout << "Index\tA\t\tB\t\tC = A + B" << std::endl;
    for (int i = 0; i < 10; ++i) {
        std::cout << i << "\t" << h_A[i] << "\t" 
                  << h_B[i] << "\t" << h_C[i] << std::endl;
    }

    // free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}

